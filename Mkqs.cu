#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include <fcntl.h>

#include <unistd.h>

#include <sys/stat.h>

#include <iostream>

#include <fstream>

#include ""

#include <stdio.h>

#include<string.h>

#include<stdlib.h>

#ifndef __HIPCC__

#define __HIPCC__

#endif

#define swap(a,b){ char *t=x[a];x[a]=x[b];x[b]=t;}

#define i2c(i)  x[i][depth]

__device__ long int min(long int a,long int b){
    return a<=b?a:b;
}

__device__ void vecswap(long int i,long int j,long int n, char *x[])
{
    while (n-- > 0) {
        swap(i, j);
        i++;
        j++;
		    }
}



__global__ void ssort1(char *x[],long int n,long int depth){
    hipStream_t s1,s2,s3;
    long int a,b,c,d,r,v;
	 if(n<=1)
	        return ;
        a=5%n;
    swap(0,a);
    v=i2c(0);
    a=b=1;
    c=d=n-1;
    for (;;)
    {
        while(b<=c && (r=i2c(b)-v)<=0){
            if (r==0) {
                swap(a,b);a++;
	              }
		  b++;
        }



        while(b<=c && (r=i2c(c)-v)>=0){
            if (r==0) {
                swap(c,d); d--;
            }
            c--;   
     }	
        if (b>c)
            break;
        swap(b,c);
        b++;

        c--;

    }

    r=min(a,b-a);

    vecswap(0,b-r,r,x);

    r = min(d-c, n-d-1);

    vecswap(b, n-r, r, x);

    r=b-a;

        hipStreamCreateWithFlags(&s1,hipStreamNonBlocking);

    ssort1<<<1,1,0,s1>>>(x,r,depth);

    if (i2c(r)!=0){

                hipStreamCreateWithFlags(&s2,hipStreamNonBlocking);

        ssort1<<< 1,1,0,s2>>>(x+r,a+n-d-1,depth+1);

        }

    r=d-c;

        hipStreamCreateWithFlags(&s3,hipStreamNonBlocking);

    ssort1<<< 1,1,0,s3>>>(x+n-r,r,depth);

}

__global__ void AddressLoader( char *d_dest, char **d_s, int size, int count)

{


        d_s[0] = &d_dest[0];

        for(long int i=0;d_dest[i]!='\0';i++)

        {
           
     if(d_dest[i] == ' '||d_dest[i] == '\n')

                {

                        d_dest[i] = '\0';

                        d_s[count++] = &d_dest[i+1];

                }

        }

}

__global__ void Printer(char **d_s, long int size)

{

        long int i;

        for(i=0;i<size;i++)

        {

                printf("%s\n", d_s[i]);

        }

}

int main(int argc,char **argv)
{
        long long int i=0,now=0;

        char *input = (char*)calloc(900000000,sizeof(char));

        int fin = open(argv[1],O_RDONLY,0);

        char buf;

        while(read(fin,&buf,sizeof(char))!=0)
        {
                input[i++] = buf;

                if(buf == ' '||buf == '\n')now++;
        }

        input[i++] = '\0';

        printf("Successful\n");

        long int size = strlen(input)+1;

        int count = 1;

        printf("Size = %ld \tWords = %lld \n", size,now);
        
	hipError_t cudaStatus;

        char *d_dest, **d_s;


        cudaStatus = hipMalloc((void**)&d_s, sizeof(char*)*now);

        if (cudaStatus != hipSuccess) {

        fprintf(stderr, "Malloc Fail\n");

    }



        cudaStatus = hipMalloc((void**)&d_dest, size);

        if (cudaStatus != hipSuccess) {

        fprintf(stderr, "Malloc Fail\n");

    }

        cudaStatus = hipMemcpy(d_dest,input, size, hipMemcpyHostToDevice);

        if (cudaStatus != hipSuccess) {

        fprintf(stderr, "Malloc Fail\n");

        }

        AddressLoader<<<1,1>>>(d_dest,d_s,size,count);

        ssort1 <<< 1,1 >>>(d_s,now,0);

        Printer<<<1,1>>>(d_s,now);

        hipDeviceSynchronize();



        //hipMemcpy(s,d_s, size, hipMemcpyDeviceToHost);

        //puts(s);

                /*int k;        

        for(k=0;s[k]!='\0';k++)

                printf(" %c \n", s[k] );

*/

        return 0;

}

